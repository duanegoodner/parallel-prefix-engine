#include "hip/hip_runtime.h"
// cuda_prefix_sum_solver.cu
//
// Defines the CUDA kernel and launch function for performing 2D prefix sum.
// This file contains only GPU-side logic and is compiled by NVCC.

#include <hip/hip_runtime.h>

#include <cstdio>
#include <iostream>

#include "cuda_prefix_sum/cuda_met_device_helpers.cuh"
#include "cuda_prefix_sum/cuda_prefix_sum_solver.cuh"
#include "cuda_prefix_sum/kernel_launch_params.hpp"


__global__ void PrefixSumKernelTiled(
    // int *d_data,
    KernelLaunchParams params
) {
  // Declare dynamic shared memory
  extern __shared__ int shared_mem[];

  // Divide shared memory into two arrays
  KernelArray array_a{.d_address = shared_mem, .size = params.array.size};

  // === Phase 1: Load input from global memory to shared memory ===
  // CopyGlobalArrayToSharedArray(params.array, array_a, params.tile_size);
  CopyMETTiledArray(params.array, array_a, params.tile_size);
  __syncthreads();

  // === Phase 2: Row-wise prefix sum within each tile of arrayA ===
  ComputeLocalRowWisePrefixSums(array_a, params.tile_size);
  __syncthreads();

  // === Phase 3: Column-wise prefix sum within each tile of arrayA ===
  ComputeLocalColWisePrefixSums(array_a, params.tile_size);
  __syncthreads();

  // === Phase 4: Broadcast right edge values to downstream elements ===
  BroadcastRightEdgesInPlace(array_a, params.tile_size);
  __syncthreads();

  // === Phase 5: Broadcast bottom edge values to downstream elements ===
  BroadcastBottomEdgesInPlace(array_a, params.tile_size);
  __syncthreads();

  // === Phase 6: Write final result back to global memory ===
  CopyMETTiledArray(array_a, params.array, params.tile_size);
}

void ConfigureSharedMemoryForKernel() {
  hipError_t err;

  // Step 1: Prefer shared memory over L1 cache
  err = hipFuncSetCacheConfig(reinterpret_cast<const void*>(
      PrefixSumKernelTiled),
      hipFuncCachePreferShared
  );
  if (err != hipSuccess) {
    fprintf(
        stderr,
        "Failed to set cache config: %s\n",
        hipGetErrorString(err)
    );
  }

  // Step 2: Request max dynamic shared memory (96 KB = 98304 bytes)
  err = hipFuncSetAttribute(reinterpret_cast<const void*>(
      PrefixSumKernelTiled),
      hipFuncAttributeMaxDynamicSharedMemorySize,
      98304
  );
  if (err != hipSuccess) {
    fprintf(
        stderr,
        "Failed to set max shared memory: %s\n",
        hipGetErrorString(err)
    );
  }
}

void LaunchPrefixSumKernelTiled(KernelLaunchParams kernel_params) {

  int num_tile_cols =
      kernel_params.array.size.num_cols / kernel_params.tile_size.num_cols;
  int num_tile_rows =
      kernel_params.array.size.num_rows / kernel_params.tile_size.num_rows;

  dim3 blockDim(num_tile_cols, num_tile_rows);
  dim3 gridDim(1, 1); // Single block for now

  int shared_mem_size = kernel_params.array.size.num_rows *
                        kernel_params.array.size.num_cols * sizeof(int);

  ConfigureSharedMemoryForKernel();

  PrefixSumKernelTiled<<<gridDim, blockDim, shared_mem_size, 0>>>(kernel_params
  );

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }

  hipDeviceSynchronize();
  hipError_t sync_err = hipGetLastError();
  if (sync_err != hipSuccess) {
    fprintf(stderr, "CUDA sync error: %s\n", hipGetErrorString(sync_err));
  }
}



