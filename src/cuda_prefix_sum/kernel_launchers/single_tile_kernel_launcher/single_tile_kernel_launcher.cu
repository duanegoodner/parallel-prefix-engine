#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <iostream>
#include <stdexcept>
#include <string>

#include "common/logger.hpp"

#include "cuda_prefix_sum/internal/kernel_config_utils.cuh"
#include "cuda_prefix_sum/internal/kernel_launch_params.hpp"
#include "cuda_prefix_sum/internal/sub_tile_kernels.cuh"
#include "cuda_prefix_sum/single_tile_kernel_launcher.cuh"

namespace sk = subtile_kernels;

SingleTileKernelLauncher::SingleTileKernelLauncher(
    const ProgramArgs &program_args
)
    : program_args_{program_args} {}

void SingleTileKernelLauncher::Launch(const KernelArray &device_array) {

  CheckProvidedTileSize();

  // Set max dynamic shared memory and prefer shared over L1
  constexpr size_t kMaxSharedMemBytes = 98304;
  ConfigureSharedMemoryForKernel(sk::SingleTileKernel, kMaxSharedMemBytes);

  // Prepare launch configuration
  dim3 block_dim = GetBlockDim();
  dim3 grid_dim = GetGridDim();
  size_t shared_mem_size = GetSharedMemSize();

  auto launch_params = CreateKernelLaunchParams(device_array, program_args_);

  // Launch the kernel
  sk::SingleTileKernel<<<grid_dim, block_dim, shared_mem_size, 0>>>(
      launch_params
  );

  // Validate
  CheckErrors();
}

dim3 SingleTileKernelLauncher::GetGridDim() { return dim3(1, 1, 1); }

dim3 SingleTileKernelLauncher::GetBlockDim() {
  if (program_args_.SubTileSize2D().num_rows == 0 ||
      program_args_.SubTileSize2D().num_cols == 0) {
    throw std::invalid_argument("Sub-tile size dimensions must be non-zero");
  }

  uint32_t num_tile_rows = static_cast<uint32_t>(
      program_args_.FullMatrixSize2D().num_rows /
      program_args_.SubTileSize2D().num_rows
  );
  uint32_t num_tile_cols = static_cast<uint32_t>(
      program_args_.FullMatrixSize2D().num_cols /
      program_args_.SubTileSize2D().num_cols
  );

  return dim3(num_tile_cols, num_tile_rows, 1);
}

size_t SingleTileKernelLauncher::GetSharedMemSize() {
  return static_cast<size_t>(program_args_.FullMatrixSize2D().num_rows) *
         static_cast<size_t>(program_args_.FullMatrixSize2D().num_cols) *
         sizeof(int);
}

void SingleTileKernelLauncher::CheckErrors() {
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }

  hipDeviceSynchronize();
  hipError_t sync_err = hipGetLastError();
  if (sync_err != hipSuccess) {
    fprintf(stderr, "CUDA sync error: %s\n", hipGetErrorString(sync_err));
  }
}

void SingleTileKernelLauncher::CheckProvidedTileSize() {
  if (program_args_.FullMatrixSize2D() != program_args_.TileSize2D()) {
    std::cout << std::endl;
    std::string tile_size =
        std::to_string(program_args_.TileSize2D().num_rows) + "x" +
        std::to_string(program_args_.TileSize2D().num_cols);
    std::string full_matrix_size =
        std::to_string(program_args_.FullMatrixSize2D().num_rows) + "x" +
        std::to_string(program_args_.FullMatrixSize2D().num_cols) + ".";
    Logger::Log(
        LogLevel::WARNING,
        "Specified tile size of " + tile_size +
            " does not match full matrix size of " + full_matrix_size
    );
    Logger::Log(
        LogLevel::WARNING,
        "Single tile kernel uses single top level tile with size equal to "
        "full matrix size."
    );
    Logger::Log(
        LogLevel::WARNING,
        "Ignoring provided tile size value of " + tile_size + "."
    );
    std::cout << std::endl;
  }
}