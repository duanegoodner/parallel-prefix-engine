#include <hip/hip_runtime.h>

#include <iostream>

#include "cuda_prefix_sum/internal/kernel_array.hpp"
#include "cuda_prefix_sum/internal/kernel_config_utils.cuh"
#include "cuda_prefix_sum/internal/kernel_launch_params.hpp"
#include "cuda_prefix_sum/internal/multi_tile_kernel.cuh"
#include "cuda_prefix_sum/multi_tile_kernel_launcher.cuh"

MultiTileKernelLauncher::MultiTileKernelLauncher(
    const ProgramArgs &program_args
)
    : program_args_{program_args}
    , right_tile_edge_buffers_{{program_args_.FullMatrixSize2D().num_rows, FirstPassGridDim().x }}
    , bottom_tile_edge_buffers_{{FirstPassGridDim().y, program_args_.FullMatrixSize2D().num_cols}} {}

void MultiTileKernelLauncher::Launch(const KernelArray &device_array) {
  constexpr size_t kMaxSharedMemBytes = 98304;
  ConfigureSharedMemoryForKernel(FirstPass, kMaxSharedMemBytes);

  // Prepare launch configuration
  dim3 block_dim = FirstPassBlockDim();
  dim3 grid_dim = FirstPassGridDim();
  size_t shared_mem_size = FirstPassSharedMemPerBlock();

  auto launch_params = CreateKernelLaunchParams(device_array, program_args_);

  FirstPass<<<grid_dim, block_dim, shared_mem_size>>>(
      launch_params,
      right_tile_edge_buffers_.View(),
      bottom_tile_edge_buffers_.View()
  );

  CheckErrors();
}

dim3 MultiTileKernelLauncher::FirstPassGridDim() {

  auto num_block_rows = program_args_.FullMatrixSize2D().num_rows /
                        program_args_.TileSize2D().num_rows;

  auto num_block_cols = program_args_.FullMatrixSize2D().num_cols /
                        program_args_.TileSize2D().num_cols;

  return dim3(num_block_cols, num_block_rows, 1);
}

dim3 MultiTileKernelLauncher::FirstPassBlockDim() {
  auto num_thread_rows = program_args_.TileSize2D().num_rows /
                         program_args_.SubTileSize2D().num_rows;

  auto num_thread_cols = program_args_.TileSize2D().num_cols /
                         program_args_.SubTileSize2D().num_cols;

  return dim3(num_thread_cols, num_thread_rows, 1);
}

size_t MultiTileKernelLauncher::FirstPassSharedMemPerBlock() {

  return static_cast<size_t>(program_args_.TileSize2D().num_rows) *
         static_cast<size_t>(program_args_.TileSize2D().num_cols) *
         sizeof(int);
}

dim3 MultiTileKernelLauncher::SecondPassGridDim() {
  
}

void MultiTileKernelLauncher::CheckErrors() {
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }

  hipDeviceSynchronize();
  hipError_t sync_err = hipGetLastError();
  if (sync_err != hipSuccess) {
    fprintf(stderr, "CUDA sync error: %s\n", hipGetErrorString(sync_err));
  }
}
