#include <hip/hip_runtime.h>

#include <iostream>

#include "common/array_size_2d.hpp"
#include "cuda_prefix_sum/internal/hillis_steele_row_kernel.cuh"
#include "cuda_prefix_sum/internal/kernel_array.hpp"
#include "cuda_prefix_sum/internal/kernel_config_utils.cuh"
#include "cuda_prefix_sum/internal/kernel_launch_params.hpp"
#include "cuda_prefix_sum/internal/multi_tile_kernel.cuh"
#include "cuda_prefix_sum/multi_tile_kernel_launcher.cuh"

MultiTileKernelLauncher::MultiTileKernelLauncher(
    const ProgramArgs &program_args
)
    : program_args_{program_args}
    , right_tile_edge_buffers_{{program_args_.FullMatrixSize2D().num_rows, FirstPassGridDim().x }}
    , right_tile_edge_buffers_ps_{{program_args_.FullMatrixSize2D().num_rows, FirstPassGridDim().x }}
    , bottom_tile_edge_buffers_{{FirstPassGridDim().y, program_args_.FullMatrixSize2D().num_cols}}
    , bottom_tile_edge_buffers_ps_{{FirstPassGridDim().y, program_args_.FullMatrixSize2D().num_cols}} {}

void MultiTileKernelLauncher::Launch(const KernelArray &device_array) {
  constexpr size_t kMaxSharedMemBytes = 98304;
  ConfigureSharedMemoryForKernel(FirstPass, kMaxSharedMemBytes);

  // Prepare launch configuration
  dim3 block_dim = FirstPassBlockDim();
  dim3 grid_dim = FirstPassGridDim();
  size_t shared_mem_size = FirstPassSharedMemPerBlock();

  auto launch_params = CreateKernelLaunchParams(device_array, program_args_);

  FirstPass<<<grid_dim, block_dim, shared_mem_size>>>(
      launch_params,
      right_tile_edge_buffers_.View(),
      bottom_tile_edge_buffers_.View()
  );

  CheckErrors();

  LaunchRowWisePrefixSum(
      right_tile_edge_buffers_.d_address(),
      right_tile_edge_buffers_ps_.d_address(),
      right_tile_edge_buffers_.size(),
      FirstPassGridDim().x
  );

  CheckErrors();
}

dim3 MultiTileKernelLauncher::FirstPassGridDim() {

  auto num_block_rows = program_args_.FullMatrixSize2D().num_rows /
                        program_args_.TileSize2D().num_rows;

  auto num_block_cols = program_args_.FullMatrixSize2D().num_cols /
                        program_args_.TileSize2D().num_cols;

  return dim3(num_block_cols, num_block_rows, 1);
}

dim3 MultiTileKernelLauncher::FirstPassBlockDim() {
  auto num_thread_rows = program_args_.TileSize2D().num_rows /
                         program_args_.SubTileSize2D().num_rows;

  auto num_thread_cols = program_args_.TileSize2D().num_cols /
                         program_args_.SubTileSize2D().num_cols;

  return dim3(num_thread_cols, num_thread_rows, 1);
}

size_t MultiTileKernelLauncher::FirstPassSharedMemPerBlock() {

  return static_cast<size_t>(program_args_.TileSize2D().num_rows) *
         static_cast<size_t>(program_args_.TileSize2D().num_cols) *
         sizeof(int);
}

void MultiTileKernelLauncher::CheckErrors() {
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }

  hipDeviceSynchronize();
  hipError_t sync_err = hipGetLastError();
  if (sync_err != hipSuccess) {
    fprintf(stderr, "CUDA sync error: %s\n", hipGetErrorString(sync_err));
  }
}

void MultiTileKernelLauncher::LaunchRowWisePrefixSum(
    const int *d_input,
    int *d_output,
    ArraySize2D size,
    int chunk_size
) {
  if (size.num_cols <= 1024) {
    dim3 block(size.num_cols);
    dim3 grid(size.num_rows);
    size_t shared_bytes = size.num_cols * sizeof(int);
    RowWiseScanSingleBlock<<<grid, block, shared_bytes>>>(
        d_input,
        d_output,
        size
    );
  } else {
    // Phase 1: scan chunks
    int num_chunks = (size.num_cols + chunk_size - 1) / chunk_size;
    dim3 grid_phase1(num_chunks, size.num_rows);
    dim3 block_phase1(chunk_size);
    size_t shared_bytes = chunk_size * sizeof(int);

    int *d_block_sums;
    hipMalloc(&d_block_sums, sizeof(int) * size.num_rows * num_chunks);

    RowWiseScanMultiBlockPhase1<<<grid_phase1, block_phase1, shared_bytes>>>(
        d_input,
        d_output,
        d_block_sums,
        size,
        chunk_size
    );

    // Phase 1.5: scan block sums
    int *d_scanned_block_sums;
    hipMalloc(&d_scanned_block_sums, sizeof(int) * size.num_rows * num_chunks);

    // Recursively scan block sums row-wise
    LaunchRowWisePrefixSum(
        d_block_sums,
        d_scanned_block_sums,
        size,
        chunk_size
    );

    // Phase 2: apply scanned sums
    RowWiseScanMultiBlockPhase2<<<grid_phase1, block_phase1>>>(
        d_output,
        d_scanned_block_sums,
        size,
        chunk_size
    );

    hipFree(d_block_sums);
    hipFree(d_scanned_block_sums);
  }
}
