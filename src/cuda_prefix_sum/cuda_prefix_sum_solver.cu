#include "hip/hip_runtime.h"
// cuda_prefix_sum_solver.cu
//
// Defines the CUDA kernel and launch function for performing 2D prefix sum.
// This file contains only GPU-side logic and is compiled by NVCC.

#include <hip/hip_runtime.h>

#include <cstdio>
#include <iostream>

#include "cuda_prefix_sum/cuda_device_helpers.cuh"
#include "cuda_prefix_sum/cuda_prefix_sum_solver.cuh"
#include "cuda_prefix_sum/kernel_launch_params.hpp"

__global__ void PrefixSumKernel(
    // int *d_data,
    KernelLaunchParams params
) {
  // Declare dynamic shared memory
  extern __shared__ int shared_mem[];

  // Divide shared memory into two arrays
  KernelArray array_a{.d_address = shared_mem, .size = params.array.size};
  KernelArray array_b{.d_address = shared_mem, .size = params.array.size};

  // === Phase 1: Load input from global memory to shared memory ===
  CopyGlobalArrayToSharedArray(params.array, array_a, params.tile_size);

  __syncthreads();

  // Debug statement: Print contents of arrayA after loading from global memory
  PrintKernelArray(
      array_a,
      "Contents of array_a after loading from global memory"
  );

  // === Phase 2: Row-wise prefix sum within each tile of arrayA ===
  for (int tile_col = 1; tile_col < params.tile_size.y; tile_col++) {
    for (int tile_row = 0; tile_row < params.tile_size.x; ++tile_row) {
      ComputeRowWisePrefixSum(array_a, params.tile_size, tile_row, tile_col);
    }
  }

  __syncthreads();
  // Debug statement: Print contents of arrayA after row-wise prefix sum
  PrintKernelArray(array_a, "Contents of array_a after row-wise prefix sum");

  // === Phase 3: Column-wise prefix sum within each tile of arrayA ===
  for (int tile_row = 1; tile_row < params.tile_size.x; tile_row++) {
    for (int tile_col = 0; tile_col < params.tile_size.y; ++tile_col) {
      int full_matrix_x = threadIdx.x * params.tile_size.x + tile_row;
      int full_matrix_y = threadIdx.y * params.tile_size.y + tile_col;
      int full_matrix_x_prev = threadIdx.x * params.tile_size.x + tile_row - 1;
      CombineElementInto(
          array_a,
          full_matrix_x_prev,
          full_matrix_y,
          full_matrix_x,
          full_matrix_y
      );
    }
  }

  __syncthreads();
  // Debug statement: Print contents of arrayA after column-wise prefix sum
  PrintKernelArray(
      array_a,
      "Contents of array_a after column-wise prefix sum"
  );

  // === Phase 4: Compute/write final result into arrayB ===

  // Extract right edges of upstream tiles
  for (int upstream_tile_col = 0; upstream_tile_col < threadIdx.y;
       ++upstream_tile_col) {
    int upstream_tile_full_matrix_col_idx =
        upstream_tile_col * params.tile_size.y + params.tile_size.y - 1;
    for (int tile_row = 0; tile_row < params.tile_size.x; ++tile_row) {
      int full_matrix_row_idx = threadIdx.x * params.tile_size.x + tile_row;
      int edge_val = array_a.d_address
                         [full_matrix_row_idx * array_a.size.y +
                          upstream_tile_full_matrix_col_idx];
      for (int tile_col = 0; tile_col < params.tile_size.y; ++tile_col) {
        int full_matrix_x = threadIdx.x * params.tile_size.x + tile_row;
        int full_matrix_y = threadIdx.y * params.tile_size.y + tile_col;
        array_b.d_address[full_matrix_x * array_b.size.y + full_matrix_y] =
            array_a.d_address[full_matrix_x * array_a.size.y + full_matrix_y] +
            edge_val;
      }
    }
  }

  __syncthreads();
  // Debug statement: Print contents of arrayB after adding upstream right
  // edges
  PrintKernelArray(
      array_b,
      "Contents of array_b extracting/adding right edges of upstream tiles"
  );

  // PrintArray(
  //     arrayB,
  //     params.arr_size_x,
  //     params.arr_size_y,
  //     "Contents of arrayB extracting/adding right edges of upstream
  //     tiles"
  // );

  // Extract bottom edges of upstream tiles

  for (int tile_row = 0; tile_row < params.tile_size.x; ++tile_row) {
    for (int tile_col = 0; tile_col < params.tile_size.y; ++tile_col) {
      int full_matrix_x = threadIdx.x * params.tile_size.x + tile_row;
      int full_matrix_y = threadIdx.y * params.tile_size.y + tile_col;
    }
  }

  // === Phase 2: Row-wise prefix sum into arrayB ===
  int sum = 0;
  for (int col = 0; col <= threadIdx.y; ++col) {
    // TODO: implement op
    sum += array_a.d_address[threadIdx.x * blockDim.y + col];
  }

  array_b.d_address[threadIdx.x * blockDim.y + threadIdx.y] = sum;

  __syncthreads();

  // PrintSharedMemoryArray(
  //     arrayB,
  //     "Contents of arrayB after row-wise prefix sum"
  // );

  // === Phase 3: Column-wise prefix sum (over arrayB) into arrayA ===
  sum = 0;
  for (int row = 0; row <= threadIdx.x; ++row) {
    // TODO: implement op
    sum += array_b.d_address[row * blockDim.y + threadIdx.y];
  }
  array_a.d_address[threadIdx.x * blockDim.y + threadIdx.y] = sum;

  __syncthreads();

  // Debug: Print contents of arrayA after column-wise prefix sum
  // PrintSharedMemoryArray(
  //     arrayA,
  //     "Contents of arrayA after column-wise prefix sum"
  // );

  // === Phase 4: Write final result back to global memory ===
  CopySharedArrayToGlobalArray(array_b, params.array, params.tile_size);

  // params.d_arr[index] = arrayA[tx * blockDim.y + ty];

  // PrintGlobalMemArray(d_data);
}

void LaunchPrefixSumKernel(
    // int *d_data,
    KernelLaunchParams kernel_params,
    hipStream_t stream
) {

  int num_tiles_x = kernel_params.array.size.x / kernel_params.tile_size.x;
  int num_tiles_y = kernel_params.array.size.y / kernel_params.tile_size.y;

  // dim3 blockDim(full_matrix_dim_x, full_matrix_dim_y);
  dim3 blockDim(num_tiles_x, num_tiles_y);
  dim3 gridDim(1, 1); // Single block for now

  PrefixSumKernel<<<gridDim, blockDim, 0, stream>>>(
      // d_data,
      kernel_params
  );

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }

  hipDeviceSynchronize();
}
