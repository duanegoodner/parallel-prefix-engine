#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cuda_prefix_sum/cuda_prefix_sum_solver.cuh"

namespace config {
    constexpr int TileDim = 32;
    constexpr int TilePitch = TileDim + 1;
    // constexpr int NumElems = TileDim * TileDim;
}

__global__ void PrefixSumKernelWarp(const int* input, int* output) {
    __shared__ int tile[config::TileDim * config::TilePitch];

    // Original thread indices
    int tx = threadIdx.x; // horizontal (column index in row-pass)
    int ty = threadIdx.y; // vertical (row index in row-pass)

    int global_idx = ty * config::TileDim + tx;
    int smem_idx   = ty * config::TilePitch + tx;

    // Load global memory into shared memory
    tile[smem_idx] = input[global_idx];
    __syncthreads();

    // --- Row-wise inclusive scan (1 warp per row) ---
    int val = tile[smem_idx];
    #pragma unroll
    for (int offset = 1; offset < config::TileDim; offset *= 2) {
        int n = __shfl_up_sync(0xffffffff, val, offset);
        if (tx >= offset) val += n;
    }
    tile[smem_idx] = val;
    __syncthreads();

    // --- Column-wise inclusive scan (1 warp per column) ---
    int row = threadIdx.x;  // Remap: threads step down a column
    int col = threadIdx.y;  // Each warp owns one column

    int smem_col_idx = row * config::TilePitch + col;
    val = tile[smem_col_idx];
    #pragma unroll
    for (int offset = 1; offset < config::TileDim; offset *= 2) {
        int n = __shfl_up_sync(0xffffffff, val, offset);
        if (row >= offset) val += n;
    }
    tile[smem_col_idx] = val;
    __syncthreads();

    // Restore thread indices to write final result
    int final_idx = ty * config::TilePitch + tx;
    output[global_idx] = tile[final_idx];
}

// Kernel launcher
void LaunchPrefixSumKernelWarp(const int* d_input, int* d_output) {
    dim3 block(config::TileDim, config::TileDim);
    PrefixSumKernelWarp<<<1, block>>>(d_input, d_output);
}
