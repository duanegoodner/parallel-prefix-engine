#include "hip/hip_runtime.h"
#include "cuda_prefix_sum/internal/device_helpers.cuh"

// === Single-block exclusive scan (Hillis-Steele) for a single row ===
__global__ void RowWiseScanSingleBlock(
    const int *__restrict__ in,
    int *__restrict__ out,
    // int num_cols
    ArraySize2D size
) {
  extern __shared__ int temp[];
  int row = blockIdx.x;
  int tid = threadIdx.x;

  if (tid >= size.num_cols)
    return;

  // Load input to shared memory
  temp[tid] = in[row * size.num_cols + tid];
  __syncthreads();

  // Inclusive Hillis-Steele scan
  for (int offset = 1; offset < size.num_cols; offset *= 2) {
    int val = 0;
    if (tid >= offset)
      val = temp[tid - offset];
    __syncthreads();
    temp[tid] += val;
    __syncthreads();
  }

  // Convert to exclusive scan
  if (tid == 0) {
    out[row * size.num_cols + tid] = 0;
  } else if (tid < size.num_cols) {
    out[row * size.num_cols + tid] = temp[tid - 1];
  }

  __syncthreads();

  KernelArrayView right_edge_buffers{const_cast<int *>(in), size};
  KernelArrayView result_array{out, size};

  if (blockIdx.x == 0 && blockIdx.y == 0) {
    PrintKernelArrayView(
        right_edge_buffers,
        "right edge buffers before row-wise prefix sum"
    );
    PrintKernelArrayView(
        result_array,
        "right edge buffers after row-wise prefix sum"
    );
  }
}

// === Multi-block (chunked) scan for long rows ===
// Phase 1: Local block scan (inclusive)
__global__ void RowWiseScanMultiBlockPhase1(
    const int *__restrict__ in,
    int *__restrict__ out,
    int *__restrict__ block_sums,
    // int num_cols,
    ArraySize2D size,
    int chunk_size
) {
  extern __shared__ int temp[];
  int row = blockIdx.y;
  int chunk_start = blockIdx.x * chunk_size;
  int tid = threadIdx.x;
  int global_idx = row * size.num_cols + chunk_start + tid;

  // Load input
  if (chunk_start + tid < size.num_cols)
    temp[tid] = in[global_idx];
  else
    temp[tid] = 0;

  __syncthreads();

  // Inclusive scan within block
  for (int offset = 1; offset < chunk_size; offset *= 2) {
    int val = 0;
    if (tid >= offset)
      val = temp[tid - offset];
    __syncthreads();
    temp[tid] += val;
    __syncthreads();
  }

  // Store scan result
  if (chunk_start + tid < size.num_cols)
    out[global_idx] = temp[tid];

  // Store block sum (last element)
  if (tid == chunk_size - 1 || chunk_start + tid == size.num_cols - 1)
    block_sums[row * gridDim.x + blockIdx.x] = temp[tid];
}

// Phase 2: Add scanned block sums to partial results
__global__ void RowWiseScanMultiBlockPhase2(
    int *__restrict__ out,
    const int *__restrict__ scanned_block_sums,
    // int num_cols,
    ArraySize2D size,
    int chunk_size
) {
  int row = blockIdx.y;
  int chunk_id = blockIdx.x;
  int tid = threadIdx.x;

  int chunk_start = chunk_id * chunk_size;
  int global_idx = row * size.num_cols + chunk_start + tid;

  int offset = scanned_block_sums[row * gridDim.x + chunk_id];
  if (chunk_id > 0 && chunk_start + tid < size.num_cols) {
    out[global_idx] += offset;
  }
}