#include "hip/hip_runtime.h"
#include "cuda_prefix_sum/internal/device_helpers.cuh"
#include "cuda_prefix_sum/internal/hillis_steele_row_kernel.cuh"

namespace dh = device_helpers;

// === Single-block exclusive scan (Hillis-Steele) for a single row ===
__global__ void RowWiseScanSingleBlock(
    const int *__restrict__ in_ptr,
    int *__restrict__ out_ptr,
    ArraySize2D size
) {
  KernelArrayViewConst in{in_ptr, size};
  KernelArrayView out{out_ptr, size};

  extern __shared__ int temp[];
  KernelArrayView shared_temp{temp, {1, size.num_cols}};

  if (dh::blockrow::ColIndex() >= size.num_cols)
    return;

  // Load input to shared memory
  dh::blockrow::LoadGlobalArrayToSharedArray(in, shared_temp, 0);
  __syncthreads();

  // Inclusive Hillis-Steele scan
  dh::blockrow::InclusiveHillsSteeleScan(shared_temp, size.num_cols);

  // Convert to exclusive scan
  dh::blockrow::ConvertInclusiveToExclusive(shared_temp, 0, out);
}

// === Multi-block (chunked) scan for long rows ===
// Phase 1: Local block scan (inclusive, then convert to exclusive)
__global__ void RowWiseScanMultiBlockPhase1(
    const int *__restrict__ in_ptr,
    int *__restrict__ out_ptr,
    int *__restrict__ block_sums_ptr,
    ArraySize2D size,
    int chunk_size
) {
  //   int row = blockIdx.y;
  int num_chunks = gridDim.x;
  int chunk_start = blockIdx.x * chunk_size;
  int col_offset = threadIdx.x;
  int global_col = chunk_start + col_offset;

  KernelArrayViewConst in{in_ptr, size};
  KernelArrayView out{out_ptr, size};

  KernelArrayView block_sums_view{
      block_sums_ptr,
      {size.num_rows, static_cast<size_t>(num_chunks)}
  };

  extern __shared__ int temp[];
  KernelArrayView shared_temp{temp, {1, static_cast<size_t>(chunk_size)}};

  // Load to shared
  if (global_col < size.num_cols) {
    shared_temp.At(0, col_offset) =
        in.At(dh::blockrow::chunks::RowIndex(), global_col);
  } else {
    shared_temp.At(0, col_offset) = 0;
  }
  __syncthreads();

  // Inclusive scan in shared memory
  for (int offset = 1; offset < chunk_size; offset *= 2) {
    int val =
        (col_offset >= offset) ? shared_temp.At(0, col_offset - offset) : 0;
    __syncthreads();
    shared_temp.At(0, col_offset) += val;
    __syncthreads();
  }

  // Convert to exclusive scan and write result
  if (global_col < size.num_cols) {
    dh::ConvertInclusiveToExclusiveRow(
        out,
        shared_temp,
        dh::blockrow::chunks::RowIndex(),
        col_offset
    );
  }

  // Store full sum for this block
  if (col_offset == chunk_size - 1 || global_col == size.num_cols - 1) {
    block_sums_view.At(
        dh::blockrow::chunks::RowIndex(),
        dh::blockrow::chunks::RowIndex()
    ) = shared_temp.At(0, col_offset);
  }
}

// Phase 2: Add scanned block sums to partial results
__global__ void RowWiseScanMultiBlockPhase2(
    int *__restrict__ out_ptr,
    const int *__restrict__ scanned_block_sums_ptr,
    ArraySize2D size,
    int chunk_size
) {
  KernelArrayView out{out_ptr, size};

  int row = blockIdx.y;
  int chunk_id = blockIdx.x;
  int col_offset = threadIdx.x;
  int chunk_start = chunk_id * chunk_size;
  int global_col = chunk_start + col_offset;

  if (chunk_id == 0 || global_col >= size.num_cols)
    return;

  int offset = scanned_block_sums_ptr[row * gridDim.x + chunk_id];
  out.At(row, global_col) += offset;
}
