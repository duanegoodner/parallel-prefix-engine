#include "hip/hip_runtime.h"


#include <hip/hip_runtime.h>

#include "cuda_prefix_sum/internal/device_helpers.cuh"
#include "cuda_prefix_sum/internal/kernel_array.hpp"
#include "cuda_prefix_sum/internal/kernel_launch_params.hpp"
#include "cuda_prefix_sum/internal/multi_tile_kernel.cuh"

__global__ void FirstPass(
    KernelLaunchParams params,
    KernelArrayView right_edges_buffer,
    KernelArrayView bottom_edges_buffer
) {
  extern __shared__ int shared_mem[];

  KernelArrayView shared_array{
      .d_address = shared_mem,
      .size = params.tile_size
  };

  CopyFromGlobalToShared(params.array, shared_array, params.sub_tile_size);
  __syncthreads();

  ComputeSharedMemArrayPrefixSum(shared_array, params.sub_tile_size);
  __syncthreads();

  CopyFromSharedToGlobal(shared_array, params.array, params.sub_tile_size);
  __syncthreads();

  CopyTileRightEdgesToGlobalBuffer(
      shared_array,
      right_edges_buffer,
      params.sub_tile_size
  );
  CopyTileBottomEdgesToGlobalBuffer(
      shared_array,
      bottom_edges_buffer,
      params.sub_tile_size
  );
  __syncthreads();

//   if (blockIdx.x == 0 && blockIdx.y == 0) {
//     PrintKernelArrayView(right_edges_buffer, "right_edges_buffer");
//     PrintKernelArrayView(bottom_edges_buffer, "bottom_edges_buffer");
//   }

  __syncthreads();
}