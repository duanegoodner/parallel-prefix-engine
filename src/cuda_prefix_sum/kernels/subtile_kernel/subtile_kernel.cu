#include "hip/hip_runtime.h"
// cuda_prefix_sum_solver.cu
//
// Defines the CUDA kernel and launch function for performing 2D prefix sum.
// This file contains only GPU-side logic and is compiled by NVCC.

#include <hip/hip_runtime.h>

#include <cstdio>
#include <iostream>

#include "cuda_prefix_sum/internal/subtile_device_helpers.cuh"
#include "cuda_prefix_sum/internal/kernel_config_utils.cuh"
#include "cuda_prefix_sum/internal/kernel_launch_params.hpp"
#include "cuda_prefix_sum/subtile_kernel_launcher.cuh"

__global__ void SubtileKernel(
    // int *d_data,
    KernelLaunchParams params
) {
  // Declare dynamic shared memory
  extern __shared__ int shared_mem[];

  // Declare shared memory
  KernelArray array_a{.d_address = shared_mem, .size = params.tile_size};
  __syncthreads();

  // === Phase 1: Load input from global memory to shared memory ===
  CopyFromGlobalToShared(params.array, array_a, params.sub_tile_size);
  __syncthreads();
  
  // === Phase 2: Compute 2D prefix sum on shared mem array ===
  ComputeSharedMemArrayPrefixSum(array_a, params.sub_tile_size);
  __syncthreads();

  // === Phase 3: Write final result back to global memory ===
  CopyFromSharedToGlobal(array_a, params.array, params.sub_tile_size);
}
