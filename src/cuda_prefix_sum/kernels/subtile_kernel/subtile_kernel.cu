#include "hip/hip_runtime.h"
// cuda_prefix_sum_solver.cu
//
// Defines the CUDA kernel and launch function for performing 2D prefix sum.
// This file contains only GPU-side logic and is compiled by NVCC.

#include <hip/hip_runtime.h>

#include <cstdio>
#include <iostream>

#include "cuda_prefix_sum/internal/subtile_device_helpers.cuh"
// #include "cuda_prefix_sum/cuda_prefix_sum_solver.cuh"
#include "cuda_prefix_sum/internal/kernel_config_utils.cuh"
#include "cuda_prefix_sum/internal/kernel_launch_params.hpp"
#include "cuda_prefix_sum/subtile_kernel_launcher.cuh"

__global__ void SubtileKernel(
    // int *d_data,
    KernelLaunchParams params
) {
  // Declare dynamic shared memory
  extern __shared__ int shared_mem[];

  // Divide shared memory into two arrays
  KernelArray array_a{.d_address = shared_mem, .size = params.array.size};

  // === Phase 1: Load input from global memory to shared memory ===
  // CopyGlobalArrayToSharedArray(params.array, array_a, params.sub_tile_size);
  CopyMETTiledArray(params.array, array_a, params.sub_tile_size);
  __syncthreads();

  // === Phase 2: Row-wise prefix sum within each tile of arrayA ===
  ComputeLocalRowWisePrefixSums(array_a, params.sub_tile_size);
  __syncthreads();

  // === Phase 3: Column-wise prefix sum within each tile of arrayA ===
  ComputeLocalColWisePrefixSums(array_a, params.sub_tile_size);
  __syncthreads();

  // === Phase 4: Broadcast right edge values to downstream elements ===
  BroadcastRightEdgesInPlace(array_a, params.sub_tile_size);
  __syncthreads();

  // === Phase 5: Broadcast bottom edge values to downstream elements ===
  BroadcastBottomEdgesInPlace(array_a, params.sub_tile_size);
  __syncthreads();

  // === Phase 6: Write final result back to global memory ===
  CopyMETTiledArray(array_a, params.array, params.sub_tile_size);
}
