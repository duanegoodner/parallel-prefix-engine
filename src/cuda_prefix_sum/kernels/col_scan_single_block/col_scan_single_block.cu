#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "common/array_size_2d.hpp"

#include "cuda_prefix_sum/internal/col_scan_single_block_kernel.cuh"
#include "cuda_prefix_sum/internal/kernel_array_view.cuh"

namespace col_scan_single_block {

  __global__ void ColScanSingleBlockKernel(
      const int *__restrict__ in_ptr,
      int *__restrict__ out_ptr,
      ArraySize2D size
  ) {
    // One thread per row, one block per column
    if (RowIndexInCol() >= size.num_rows)
      return;

    KernelArrayViewConst in{in_ptr, size};
    KernelArrayView out{out_ptr, size};

    extern __shared__ int temp[];
    KernelArrayView shared_temp{temp, {size.num_rows, 1}};

    LoadGlobalToSharedColumn(in, shared_temp);
    __syncthreads();

    InclusiveScanDownColumn(shared_temp, size.num_rows);
    __syncthreads();

    ConvertToExclusiveAndStore(shared_temp, out);
  }

} // namespace col_scan_single_block
